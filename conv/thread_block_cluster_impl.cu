#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <ctime>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_runtime_api.h> 

void checkCudaErrors(hipError_t err) 
{
    if (err!= hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }
}

__global__ void convHalfKernel(half *input, half *output, int kernelSize, int stride, int inChannels, int imageHeight, int imageWidth)
{

    int globalRow = blockIdx.y * blockDim.y + threadIdx.y;
    int globalCol = blockIdx.x * blockDim.x + threadIdx.x;


    if (globalRow < imageHeight && globalCol < imageWidth) {
        half sum = 0;
        for (int inChannel = 0; inChannel < inChannels; ++inChannel) {
            for (int kRow = 0; kRow < kernelSize; ++kRow) {
                for (int kCol = 0; kCol < kernelSize; ++kCol) {
                    int inputRow = globalRow + kRow * stride;
                    int inputCol = globalCol + kCol * stride;

                    if (inputRow >= 0 && inputRow < imageHeight && inputCol >= 0 && inputCol < imageWidth) {
                        sum += input[(inChannel * imageHeight + inputRow) * imageWidth + inputCol] * 
                               kernel[(inChannel * kernelSize * kernelSize) + (kRow * kernelSize + kCol)];
                    }
                }
            }
        }
        output[(globalRow * imageWidth + globalCol)] = sum;
    }
}

int main()
{
    int kernelSize = 3;
    int stride = 1;
    int inChannels = 3;
    int imageHeight = 128;
    int imageWidth = 128;


    half *h_input, *h_output;
    size_t inputSize = imageHeight * imageWidth * inChannels * sizeof(half);
    size_t outputSize = imageHeight * imageWidth * sizeof(half);

    h_input = (half *)malloc(inputSize);
    h_output = (half *)malloc(outputSize);


    std::srand(static_cast<unsigned int>(std::time(nullptr)));
    for (size_t i = 0; i < inputSize; ++i) {
        h_input[i] = static_cast<half>(std::rand() / static_cast<float>(RAND_MAX));
    }


    half *d_input, *d_output;
    hipError_t err = hipMalloc(&d_input, inputSize);
    checkCudaErrors(err);
    err = hipMalloc(&d_output, outputSize);
    checkCudaErrors(err);


    err = hipMemcpy(d_input, h_input, inputSize, hipMemcpyHostToDevice);
    checkCudaErrors(err);

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((imageWidth + threadsPerBlock.x - 1) / threadsPerBlock.x, (imageHeight + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // with thread block cluster, kernel launch!
    {
        hipLaunchConfig_t config = {0};
        config.gridDim = numBlocks;
        config.blockDim = threadsPerBlock;

        hipLaunchAttribute attribute[1];
        attribute[0].id = cudaLaunchAttributeClusterDimension;
        attribute[0].val.clusterDim.x = 4; 
        attribute[0].val.clusterDim.y = 4;
        attribute[0].val.clusterDim.z = 1;
        config.attrs = attribute;
        config.numAttrs = 1;

        cudaLaunchKernelEx(&config, convHalfKernel, d_input, d_output, kernelSize, stride, inChannels, imageHeight, imageWidth);
    }

    err = hipMemcpy(h_output, d_output, outputSize, hipMemcpyDeviceToHost);
    checkCudaErrors(err);

    for (int i = 0; i < 10; ++i) {
        std::cout << static_cast<float>(h_output[i]) << " ";
    }
    std::cout << std::endl;

    err = hipFree(d_input);
    checkCudaErrors(err);
    err = hipFree(d_output);
    checkCudaErrors(err);
    free(h_input);
    free(h_output);

    return 0;
}
