#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <ctime>
#include <hip/hip_runtime.h>

void checkCudaErrors(hipError_t err) {
    if (err!= hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }
}

// 对图片矩阵进行归一化（减均值除以方差）
__global__ void NormalizeImage(float *d_imageData, float d_mean, float d_scale, int d_numPixels, float *d_normalizeExecutionTime) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < d_numPixels) {
        d_imageData[index] = (d_imageData[index] - d_mean) / d_scale;
    }
    hipEventRecord(*d_normalizeExecutionTime);
}

// 对图片矩阵进行转置
__global__ void TransposeImage(float *d_input, float *d_output, int d_width, int d_height, float *d_transposeExecutionTime) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < d_width && y < d_height) {
        d_output[y * d_width + x] = d_input[x * d_height + y];
    }
    hipEventRecord(*d_transposeExecutionTime);
}

// 对两个矩阵进行点乘
__global__ void DotProduct(float *d_matrix1, float *d_matrix2, float *d_result, int d_numPixels) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < d_numPixels) {
        d_result[index] = d_matrix1[index] * d_matrix2[index];
    }
}

int main() {
    int d_width = 1024;
    int d_height = 1024;
    int d_numPixels = d_width * d_height;

    float *h_imageData, *h_transposedImage, *h_dotProductResult;
    float *d_imageData, *d_transposedImage, *d_dotProductResult;

    // 分配cpu内存
    h_imageData = (float *)malloc(d_numPixels * sizeof(float));
    h_transposedImage = (float *)malloc(d_numPixels * sizeof(float));
    h_dotProductResult = (float *)malloc(d_numPixels * sizeof(float));

    // 随机初始化图片矩阵
    for (int i = 0; i < d_numPixels; ++i) {
        h_imageData[i] = static_cast<float>(rand()) / RAND_MAX;
    }

    // 在gpu上分配内存
    checkCudaErrors(hipMalloc(&d_imageData, d_numPixels * sizeof(float)));
    checkCudaErrors(hipMalloc(&d_transposedImage, d_numPixels * sizeof(float)));
    checkCudaErrors(hipMalloc(&d_dotProductResult, d_numPixels * sizeof(float)));

    // h2d
    checkCudaErrors(hipMemcpy(d_imageData, h_imageData, d_numPixels * sizeof(float), hipMemcpyHostToDevice));

    float d_mean = 128.0f;  
    float d_scale = 0.4918f;  

    dim3 threadsPerBlock(256);
    dim3 numBlocks(d_numPixels / threadsPerBlock.x);

    // 记录kernel执行时间的事件
    hipEvent_t d_normalizeTime, d_transposeTime, d_dotProductTime;
    checkCudaErrors(hipEventCreate(&d_normalizeTime));
    checkCudaErrors(hipEventCreate(&d_transposeTime));
    checkCudaErrors(hipEventCreate(&d_dotProductTime));

    // 异步launch 归一化
    hipStream_t d_normalizeStream;
    checkCudaErrors(hipStreamCreate(&d_normalizeStream));
    float *d_normalizeExecutionTime;
    checkCudaErrors(hipMalloc(&d_normalizeExecutionTime, sizeof(float)));
    checkCudaErrors(NormalizeImage<<<numBlocks, threadsPerBlock, 0, d_normalizeStream>>>(d_imageData, d_mean, d_scale, d_numPixels, d_normalizeExecutionTime));

    // 异步launch 转置
    hipStream_t d_transposeStream;
    checkCudaErrors(hipStreamCreate(&d_transposeStream));
    float *d_transposeExecutionTime;
    checkCudaErrors(hipMalloc(&d_transposeExecutionTime, sizeof(float)));
    checkCudaErrors(TransposeImage<<<numBlocks, threadsPerBlock, 0, d_transposeStream>>>(d_imageData, d_transposedImage, d_width, d_height, d_transposeExecutionTime));

    // 等待两个异步操作结束
    checkCudaErrors(hipStreamSynchronize(d_normalizeStream));
    checkCudaErrors(hipStreamSynchronize(d_transposeStream));

    dim3 threadsPerBlockDotProduct(256);
    dim3 numBlocksDotProduct(d_numPixels / threadsPerBlockDotProduct.x);

    // 拿取结果进行点乘
    hipStream_t d_dotProductStream;
    checkCudaErrors(hipStreamCreate(&d_dotProductStream));
    checkCudaErrors(DotProduct<<<numBlocksDotProduct, threadsPerBlockDotProduct, 0, d_dotProductStream>>>(d_imageData, d_transposedImage, d_dotProductResult, d_numPixels));

    checkCudaErrors(hipEventRecord(d_dotProductTime));
    checkCudaErrors(hipStreamSynchronize(d_dotProductStream));

    // d2h
    checkCudaErrors(hipMemcpy(h_imageData, d_imageData, d_numPixels * sizeof(float), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(h_transposedImage, d_transposedImage, d_numPixels * sizeof(float), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(h_dotProductResult, d_dotProductResult, d_numPixels * sizeof(float), hipMemcpyDeviceToHost));

    // 计算耗时
    float normalizeElapsedTime, transposeElapsedTime, dotProductElapsedTime;
    checkCudaErrors(hipEventElapsedTime(&normalizeElapsedTime, d_normalizeTime, d_normalizeExecutionTime));
    checkCudaErrors(hipEventElapsedTime(&transposeElapsedTime, d_transposeTime, d_transposeExecutionTime));
    checkCudaErrors(hipEventElapsedTime(&dotProductElapsedTime, d_dotProductTime, NULL));

    std::cout << "Normalize time: " << normalizeElapsedTime << " ms" << std::endl;
    std::cout << "Transpose time: " << transposeElapsedTime << " ms" << std::endl;
    std::cout << "Dot Product time: " << dotProductElapsedTime << " ms" << std::endl;

    // 释放资源
    checkCudaErrors(hipFree(d_imageData));
    checkCudaErrors(hipFree(d_transposedImage));
    checkCudaErrors(hipFree(d_dotProductResult));
    checkCudaErrors(hipFree(d_normalizeExecutionTime));
    checkCudaErrors(hipFree(d_transposeExecutionTime));
    free(h_imageData);
    free(h_transposedImage);
    free(h_dotProductResult);
    checkCudaErrors(hipEventDestroy(d_normalizeTime));
    checkCudaErrors(hipEventDestroy(d_transposeTime));
    checkCudaErrors(hipEventDestroy(d_dotProductTime));
    checkCudaErrors(hipStreamDestroy(d_normalizeStream));
    checkCudaErrors(hipStreamDestroy(d_transposeStream));
    checkCudaErrors(hipStreamDestroy(d_dotProductStream));

    // 检查 CUDA error
    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus!= hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(cudaStatus) << std::endl;
        return 1;
    }

    hipDeviceReset();

    return 0;
}