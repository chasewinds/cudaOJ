#include <iostream>
#include <cstdlib>
#include <ctime>
#include <hip/hip_runtime.h>

// 矩阵点乘
__global__ void matrixDotProduct(int8_t *matrixA, int8_t *matrixB, int8_t *result, int numRows, int numCols) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;

    if (index < numRows * numCols) {
        int row = index / numCols;
        int col = index % numCols;
        result[index] = matrixA[index] * matrixB[index];
    }
}

void checkCudaErrors(hipError_t err) {
    if (err!= hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }
}

int main() {
    int numRows = 128;  
    int numCols = 128;  

    size_t matrixSize = numRows * numCols * sizeof(int8_t);

    // 在cpu上分配内存
    int8_t *h_matrixA, *h_matrixB, *h_result;
    h_matrixA = (int8_t *)malloc(matrixSize);
    h_matrixB = (int8_t *)malloc(matrixSize);
    h_result = (int8_t *)malloc(matrixSize);

    // 随机初始化矩阵数据
    std::srand(static_cast<unsigned int>(std::time(nullptr)));
    for (size_t i = 0; i < matrixSize; ++i) {
        h_matrixA[i] = static_cast<int8_t>(std::rand() % 10);
        h_matrixB[i] = static_cast<int8_t>(std::rand() % 10);
    }

    // 在gpu上分配内存
    int8_t *d_matrixA, *d_matrixB, *d_result;
    checkCudaErrors(hipMalloc(&d_matrixA, matrixSize));
    checkCudaErrors(hipMalloc(&d_matrixB, matrixSize));
    checkCudaErrors(hipMalloc(&d_result, matrixSize));

    checkCudaErrors(hipMemcpy(d_matrixA, h_matrixA, matrixSize, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_matrixB, h_matrixB, matrixSize, hipMemcpyHostToDevice));

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((numCols + threadsPerBlock.x - 1) / threadsPerBlock.x, (numRows + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // kernel launch！
    matrixDotProduct<<<numBlocks, threadsPerBlock>>>(d_matrixA, d_matrixB, d_result, numRows, numCols);

    checkCudaErrors(hipGetLastError());

    // d2h
    checkCudaErrors(hipMemcpy(h_result, d_result, matrixSize, hipMemcpyDeviceToHost));

    std::cout << "Matrix Dot Product Result:" << std::endl;
    for (int i = 0; i < numRows; ++i) {
        for (int j = 0; j < numCols; ++j) {
            std::cout << static_cast<int>(h_result[i * numCols + j]) << " ";
        }
        std::cout << std::endl;
    }

    // 释放资源
    checkCudaErrors(hipFree(d_matrixA));
    checkCudaErrors(hipFree(d_matrixB));
    checkCudaErrors(hipFree(d_result));
    free(h_matrixA);
    free(h_matrixB);
    free(h_result);

    return 0;
}